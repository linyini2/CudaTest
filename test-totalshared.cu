#include "hip/hip_runtime.h"
#include "error.cuh"
#include <stdio.h>
#include <iostream>
using namespace std;

#ifdef USE_DP
    typedef float real;
#else
    typedef long long real;
#endif

// #CASE 2:
// typedef long long real, BLOCK_SIZE = 78, GRID_SIZE = 400000
// number of threads: 31200000, total share memory: 18.1317G, share memory per block: 47.5312K
// ？？BLOCK_SIZE must be power of 2, or else the sum will be wrong

const int BLOCK_SIZE = 78;
const long long GRID_SIZE = 6000000;
const long long N = BLOCK_SIZE * GRID_SIZE;
const long long M = sizeof(real) * N;
void __global__ reduce_shared(real *d_x, real *d_y);


int main(void)
{
    real *h_x = (real *) malloc(M);
    for (long long n = 0; n < N; ++n)
    {
        h_x[n] = 1;
    }
    real *d_x;
    CHECK(hipMalloc(&d_x, M));
    CHECK(hipMemcpy(d_x, h_x, M, hipMemcpyHostToDevice));

    const long long ymem = sizeof(real) * GRID_SIZE;
    real *d_y;
    CHECK(hipMalloc(&d_y, ymem));
    real *h_y = (real *) malloc(ymem);

    reduce_shared<<<GRID_SIZE, BLOCK_SIZE>>>(d_x, d_y);

    CHECK(hipMemcpy(h_y, d_y, ymem, hipMemcpyDeviceToHost));
    cout << "number of threads: " << N << endl;
    cout << "total shared memory: " << (double)GRID_SIZE * BLOCK_SIZE * BLOCK_SIZE * sizeof(real) / (1024 * 1024 * 1024) << "G" << endl;
    cout << "shared memory per block: " << (double)BLOCK_SIZE * BLOCK_SIZE * sizeof(real) / 1024 << "K" << endl;

    

    real result = 0.0;
    for (long long n = 0; n < GRID_SIZE; ++n)
    {
        result += h_y[n];
    }
    cout << result << endl;
    // printf("%.2f", result);
    
    free(h_x);
    CHECK(hipFree(d_x));
    free(h_y);
    CHECK(hipFree(d_y));
    return 0;
}


void __global__ reduce_shared(real *d_x, real *d_y)
{
    const int tid = threadIdx.x;
    const int bid = blockIdx.x;
    const long long n = bid * blockDim.x + tid;
    __shared__ real s_y[BLOCK_SIZE][BLOCK_SIZE];
    for (int i = 0; i < BLOCK_SIZE; i++)
    {
        s_y[tid][i] = (n < N) ? d_x[n] : 0.0;
    }
    
    // s_y[tid][0] = (n < N) ? d_x[n] : 0.0;
    
    __syncthreads();

    for (int offset = blockDim.x >> 1; offset > 0; offset >>= 1)
    {

        if (tid < offset)
        {
            s_y[tid][0] += s_y[tid + offset][0];
        }
        __syncthreads();
    }

    if (tid == 0)
    {
        d_y[bid] = s_y[0][0];
    }
}
